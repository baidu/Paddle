#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/clip_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/cast_kernel.h"
#include "paddle/phi/kernels/expand_kernel.h"
#include "paddle/phi/kernels/funcs/broadcast_function.h"
#include "paddle/phi/kernels/funcs/elementwise_functor.h"

namespace phi {

template <typename T>
struct ClipTensorFunctor {
  inline HOSTDEVICE T operator()(const T x, const T min_, const T max_) const {
    return x < min_ ? min_ : x > max_ ? max_ : x;
  }
};

template <typename T, typename Context>
void ClipTensorKernel(const Context& dev_ctx,
                      const DenseTensor& x,
                      const DenseTensor& min,
                      const DenseTensor& max,
                      DenseTensor* out) {
  DenseTensor ex_min;
  DenseTensor ex_max;
  DenseTensor ex_x;
  std::vector<int> real_target_shape = common::vectorize<int>(out->dims());
  if (x.dims() != out->dims()) {
    phi::ExpandKernel<T, Context>(dev_ctx, x, real_target_shape, &ex_x);
  } else {
    ex_x = x;
  }
  if (min.dims() != out->dims()) {
    phi::ExpandKernel<T, Context>(dev_ctx, min, real_target_shape, &ex_min);
  } else {
    ex_min = min;
  }
  if (max.dims() != out->dims()) {
    phi::ExpandKernel<T, Context>(dev_ctx, max, real_target_shape, &ex_max);
  } else {
    ex_max = max;
  }
  phi::CastKernel<T, Context>(dev_ctx, ex_min, ex_x.dtype(), &ex_min);
  phi::CastKernel<T, Context>(dev_ctx, ex_max, ex_x.dtype(), &ex_max);

  std::vector<const DenseTensor*> ins = {&ex_x, &ex_min, &ex_max};
  std::vector<DenseTensor*> outs = {out};
  dev_ctx.template Alloc<T>(out);

  ClipTensorFunctor<T> func;
  funcs::ElementwiseKernel<T, ClipTensorFunctor<T>, 1>(
      dev_ctx, ins, &outs, func);
}

}  // namespace phi

PD_REGISTER_KERNEL(clip_tensor,
                   GPU,
                   ALL_LAYOUT,
                   phi::ClipTensorKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
