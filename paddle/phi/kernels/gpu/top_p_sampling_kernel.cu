#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/top_p_sampling_kernel.h"

#ifdef PADDLE_WITH_HIP
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hiprand_kernel.h>
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#else
#include <hip/hip_fp16.h>
#include <hiprand/hiprand_kernel.h>
#include <hipcub/hipcub.hpp>
#endif

#if defined(__HIPCC__) && CUDA_VERSION >= 11060
#define CUDA_BFLOAT16_AVAILABLE
#include <hip/hip_bf16.h>
#endif

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_device_function.h"

#include "paddle/common/flags.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/gather.cu.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/top_k_function_cuda.h"
#include "paddle/phi/kernels/primitive/functor_primitives.h"

#ifdef PADDLE_WITH_CUDA
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#endif

#ifdef PADDLE_WITH_HIP
#define GPU(str) hip##str
#else
#define GPU(str) cu##str
#endif

PHI_DECLARE_bool(use_air_topp);

namespace phi {

#if defined(PADDLE_WITH_CUDA) && CUDA_VERSION >= 12000
template <typename T, typename IdxT = int, typename AccT = T>
struct alignas(128) Counter {
  T const* in;
  IdxT const* inIdx;

  IdxT oriLen;

  AccT sum;
  IdxT len;
  float p;

  IdxT previousLen;

  typename hipcub::Traits<T>::UnsignedBits kthValueBits;

  alignas(128) IdxT filterCnt;

  alignas(128) uint32_t finishedBlockCnt;
};

template <typename IntType>
constexpr __host__ __device__ IntType ceilDiv(IntType a, IntType b) {
  return (a + b - 1) / b;
}

template <typename IntType>
constexpr __host__ __device__ IntType alignTo(IntType a, IntType b) {
  return ceilDiv(a, b) * b;
}

/**
 * This function calculate the bufLen, which is the size of buffer.
 * When the number of candidates for next pass exceeds the bufLen, we choose not
 * to store the candidates. Otherwise, we will load candidates from the original
 * input data.
 */
template <typename T, typename IdxT>
__host__ __device__ IdxT calcBufLen(IdxT len) {
  IdxT constexpr ratio = 2 + sizeof(IdxT) * 2 / sizeof(T);

  IdxT bufLen = len / (ratio * 8);
  bufLen = alignTo(bufLen, 256);
  return bufLen;
}

template <typename T, int BitsPerPass>
__host__ __device__ constexpr int calcNumPasses() {
  return ceilDiv<int>(sizeof(T) * 8, BitsPerPass);
}

template <typename T>
__device__ typename hipcub::Traits<T>::UnsignedBits twiddleIn(T key,
                                                           bool selectMin) {
  auto bits = reinterpret_cast<typename hipcub::Traits<T>::UnsignedBits&>(key);
  bits = hipcub::Traits<T>::TwiddleIn(bits);
  if (!selectMin) {
    bits = ~bits;
  }
  return bits;
}

template <typename T>
__device__ T twiddleOut(typename hipcub::Traits<T>::UnsignedBits bits,
                        bool selectMin) {
  if (!selectMin) {
    bits = ~bits;
  }
  bits = hipcub::Traits<T>::TwiddleOut(bits);
  return reinterpret_cast<T&>(bits);
}

template <int BitsPerPass>
__host__ __device__ constexpr int calcNumBuckets() {
  return 1 << BitsPerPass;
}

template <typename T, int BitsPerPass, int Pass>
__device__ constexpr int calcStartBit() {
  constexpr int tmpBit = sizeof(T) * 8 - (Pass + 1) * BitsPerPass;

  constexpr int startBit = tmpBit < 0 ? 0 : tmpBit;
  return startBit;
}

template <typename T, int BitsPerPass, int Pass>
__device__ constexpr uint32_t calcMask() {
  static_assert(BitsPerPass <= 31);
  constexpr int numBits = calcStartBit<T, BitsPerPass, Pass - 1>() -
                          calcStartBit<T, BitsPerPass, Pass>();
  return (1 << numBits) - 1;
}

/**
 * Find the bucket based on the radix
 */
template <typename T, int BitsPerPass>
__device__ int calcBucket(T x, int startBit, uint32_t mask, bool selectMin) {
  return (twiddleIn(x, selectMin) >> startBit) & mask;
}

/**
 *  Replace histogram with its own prefix sum (step 2 in `airTopPSampling`
 * description)
 */
template <typename IdxT, int BitsPerPass, int BlockSize>
__device__ void scan(IdxT volatile* histogram, IdxT* histogramOut) {
  int constexpr numBuckets = calcNumBuckets<BitsPerPass>();
  if constexpr (numBuckets >= BlockSize) {
    static_assert(numBuckets % BlockSize == 0);
    int constexpr itemsPerThread = numBuckets / BlockSize;
    typedef cub::
        BlockLoad<IdxT, BlockSize, itemsPerThread, hipcub::BLOCK_LOAD_TRANSPOSE>
            BlockLoad;
    typedef cub::
        BlockStore<IdxT, BlockSize, itemsPerThread, hipcub::BLOCK_STORE_TRANSPOSE>
            BlockStore;
    typedef hipcub::BlockScan<IdxT, BlockSize> BlockScan;

    __shared__ union {
      typename BlockLoad::TempStorage load;
      typename BlockScan::TempStorage scan;
      typename BlockStore::TempStorage store;
    } tempStorage;

    IdxT threadData[itemsPerThread];  // NOLINT

    BlockLoad(tempStorage.load).Load(histogram, threadData);
    __syncthreads();

    BlockScan(tempStorage.scan).InclusiveSum(threadData, threadData);
    __syncthreads();

    BlockStore(tempStorage.store).Store(histogramOut, threadData);
  } else {
    typedef hipcub::BlockScan<IdxT, BlockSize> BlockScan;
    __shared__ typename BlockScan::TempStorage tempStorage;

    IdxT threadData = 0;
    if (threadIdx.x < numBuckets) {
      threadData = histogram[threadIdx.x];
    }

    BlockScan(tempStorage).InclusiveSum(threadData, threadData);
    __syncthreads();

    if (threadIdx.x < numBuckets) {
      histogramOut[threadIdx.x] = threadData;
    }
  }
}

template <typename T, int BitsPerPass, int NumBuckets, int Pass>
__device__ __forceinline__ void filterAndHistogram(const T* in_buffer,
                                                   const int* in_idx_buffer,
                                                   T* out_buffer,
                                                   int* out_idx_buffer,
                                                   T* out_scores,
                                                   int64_t* out_ids,
                                                   int previous_len,
                                                   Counter<T>* counter,
                                                   T* histogram,
                                                   int* count_histogram,
                                                   T* histogram_shm,
                                                   int* count_histogram_shm,
                                                   const bool early_stop) {
  // scan and filter
  constexpr int start_bit = calcStartBit<T, BitsPerPass, Pass>();
  const uint32_t mask = calcMask<T, BitsPerPass, Pass>();
  constexpr int VecSize = 16 / sizeof(T);
  const int bid = blockIdx.y, tid = threadIdx.x;
  using VecT = uint4;
  union {
    VecT v;
    T array[VecSize];
  } vec;
  for (int i = (blockIdx.x * blockDim.x + threadIdx.x);
       i < ceilDiv(previous_len, VecSize);
       i += blockDim.x * gridDim.x) {
    vec.v = reinterpret_cast<const VecT*>(in_buffer)[i];
    if constexpr (Pass == 0) {
#pragma unroll
      for (int j = 0; j < VecSize; j++) {
        if (i * VecSize + j < previous_len) {
          int bucket =
              calcBucket<T, BitsPerPass>(vec.array[j], start_bit, mask, false);
          atomicAdd(histogram_shm + bucket, vec.array[j]);
          atomicAdd(count_histogram_shm + bucket, 1);
        }
      }
    } else {
      int* filter_cnt = &counter->filterCnt;
      const auto kthValueBits = counter->kthValueBits;
      constexpr int previousStartBit = calcStartBit<T, BitsPerPass, Pass - 1>();
#pragma unroll
      for (int j = 0; j < VecSize; j++) {
        const int idx = i * VecSize + j;
        if (idx < previous_len) {
          const auto previousBits =
              (twiddleIn(vec.array[j], false) >> previousStartBit)
              << previousStartBit;
          if (previousBits == kthValueBits) {
            if (early_stop) {
              const int pos = in_idx_buffer ? in_idx_buffer[idx] : idx;
              out_scores[bid] = vec.array[j];
              out_ids[bid] = pos;
            }
            if (out_buffer) {
              int pos = atomicAdd(filter_cnt, 1);
              out_buffer[pos] = vec.array[j];
              out_idx_buffer[pos] = in_idx_buffer ? in_idx_buffer[idx] : idx;
            }
            int bucket = calcBucket<T, BitsPerPass>(
                vec.array[j], start_bit, mask, false);
            atomicAdd(histogram_shm + bucket, vec.array[j]);
            atomicAdd(count_histogram_shm + bucket, 1);
          }
        }
      }
    }
  }
  __syncthreads();
  if (early_stop) {
    return;
  }
  // 合并多个block的结果
  for (int i = tid; i < NumBuckets; i += blockDim.x) {
    if (count_histogram_shm[i] > 0) {
      atomicAdd(histogram + i, histogram_shm[i]);
      atomicAdd(count_histogram + i, count_histogram_shm[i]);
    }
  }
}

#define BID 106
#define BATCH_ID 1

template <typename T, int BitsPerPass, int BlockSize, int NumBuckets, int Pass>
__global__ void air_topp_sampling(Counter<T>* counters,
                                  T* histograms,
                                  int* count_histograms,
                                  T* out,
                                  int64_t* ids,
                                  T* buf1,
                                  int* idx_buf1,
                                  T* buf2,
                                  int* idx_buf2,
                                  int* count_iter,
                                  int* count_iter_begin,
                                  const int buf_len) {
  /***
   * calc - filter - scan -find
   * TODO: calc - scan - find - filter
   ***/
  const int bid = blockIdx.y;
  if (count_iter_begin[bid] == count_iter[bid + 1]) {
    // topk
    return;
  }

  const int tid = threadIdx.x;
  auto counter = counters + bid;

  T current_sum;
  int previous_len, current_len;
  if constexpr (Pass == 0) {
    current_sum = 0;
    previous_len = counter->len;
    current_len = counter->len;
  } else {
    current_sum = counter->sum;
    previous_len = counter->previousLen;
    current_len = counter->len;
  }
  if (current_len == 0) {
    return;
  }
  const bool early_stop = (current_len == 1);
  const T* in_buf = nullptr;
  const int* in_idx_buf = nullptr;
  T* out_buf = nullptr;
  int* out_idx_buf = nullptr;
  const int buf_offset = bid * buf_len;
  if constexpr (Pass == 0) {
    in_buf = counter->in;
    in_idx_buf = nullptr;
    out_buf = nullptr;
    out_idx_buf = nullptr;
  } else if constexpr (Pass == 1) {
    in_buf = counter->in;
    in_idx_buf = nullptr;
    out_buf = buf1 + buf_offset;
    out_idx_buf = idx_buf1 + buf_offset;
  } else {
    in_buf = buf1 + buf_offset;
    in_idx_buf = idx_buf1 + buf_offset;
    out_buf = buf2 + buf_offset;
    out_idx_buf = idx_buf2 + buf_offset;
  }

  if (Pass == 0 || Pass == 1 || previous_len > buf_len) {
    // 没有写入buffer，滞后一个pass
    // 表示上一轮没有写入buf
    previous_len = counter->oriLen;
    in_buf = counter->in;
    in_idx_buf = nullptr;
  }
  if (Pass == 0 || current_len > buf_len) {
    // 当前pass无需写入buffer
    out_buf = nullptr;
    out_idx_buf = nullptr;
  }

#ifdef DEBUG_TOPP
  if (blockIdx.x == BID && bid == BATCH_ID && tid == 0) {
    printf("previous_len: %d, current_len: %d, buf_len: %d, NumBuckets: %d\n",
           previous_len,
           current_len,
           buf_len,
           NumBuckets);
  }
  __syncthreads();
#endif

  auto histogram = histograms + bid * NumBuckets;
  auto count_histogram = count_histograms + bid * NumBuckets;
  __shared__ T histogram_shm[NumBuckets];
  __shared__ int count_histogram_shm[NumBuckets];
  for (int i = tid; i < NumBuckets; i += blockDim.x) {
    histogram_shm[i] = 0;
    count_histogram_shm[i] = 0;
  }
  __syncthreads();

  filterAndHistogram<T, BitsPerPass, NumBuckets, Pass>(in_buf,
                                                       in_idx_buf,
                                                       out_buf,
                                                       out_idx_buf,
                                                       out,
                                                       ids,
                                                       previous_len,
                                                       counter,
                                                       histogram,
                                                       count_histogram,
                                                       histogram_shm,
                                                       count_histogram_shm,
                                                       early_stop);
  __syncthreads();
  // 保证全局内存操作对所有grid可见
  __threadfence();

  // #ifdef DEBUG_TOPP
  //   if (blockIdx.x == BID && bid == 0 && tid == 0) {
  //     for (int i = 0; i < NumBuckets; ++i) {
  //       printf("histogram[%d]: %f, count_histogram[%d]: %d\n", i,
  //       histogram[i], i, count_histogram[i]);
  //     }
  //   }
  //   __syncthreads();
  // #endif

  // find last block
  bool isLastBlock = false;
  if (threadIdx.x == 0) {
    uint32_t finished = atomicInc(&counter->finishedBlockCnt, gridDim.x - 1);
    isLastBlock = (finished == (gridDim.x - 1));
  }

  if (__syncthreads_or(isLastBlock)) {
    if (early_stop) {
      if (threadIdx.x == 0) {
        counter->previousLen = 0;
        counter->len = 0;
      }
      return;
    }

    // scan/find
    // constexpr int WARP_SIZE = 32;
    constexpr int WARP_COUNT = NumBuckets / WARP_SIZE;
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    __shared__ T warpSum[WARP_COUNT];
    __shared__ cuda::atomic<T, cuda::thread_scope_block> blockSum;
    for (int i = tid; i < WARP_COUNT; i += BlockSize) {
      warpSum[i] = 0;
    }
    if (tid == 0) {
      blockSum = 0;
    }
    __syncthreads();
    // Acquire the summation of each 32 buckets
    for (int i = threadIdx.x; i < NumBuckets; i += BlockSize) {
      reduce_store_async(
          warp, warpSum + i / WARP_SIZE, histogram[i], cg::plus<float>{});
    }
    __syncthreads();
    // Acquire the summation of all the 2048 buckets
    if (threadIdx.x < WARP_SIZE) {
      reduce_store_async(
          warp, blockSum, warpSum[threadIdx.x], cg::plus<float>{});
      reduce_update_async(
          warp, blockSum, warpSum[threadIdx.x + WARP_SIZE], cg::plus<float>{});
    }
    __syncthreads();

    if constexpr (Pass == 0) {
      current_sum = blockSum * counter->p;
    }

    if (tid == 0) {
      T prev = 0;

      // Add 32 elements each step
      int iStep = 0;
      int targetStep = 0;
      for (; iStep < WARP_COUNT; iStep++) {
        if (warpSum[iStep]) {
          targetStep = iStep;
          if ((prev + warpSum[iStep]) >= current_sum) {
            break;
          }
          prev += warpSum[iStep];
        }
      }

      int targetIdx = 0;
      for (int i = targetStep * WARP_SIZE; i < NumBuckets; i++) {
        if (count_histogram[i]) {
          targetIdx = i;
          if ((prev + histogram[i]) >= current_sum) {
            break;
          }
          prev += histogram[i];
        }
      }
      counter->sum =
          current_sum - prev;  // how many values still are there to find
      counter->len = count_histogram[targetIdx];  // cur - prev; // number of
                                                  // values in next pass
      typename hipcub::Traits<T>::UnsignedBits bucket = targetIdx;
      int startBit = calcStartBit<T, BitsPerPass, Pass>();
      counter->kthValueBits |= bucket << startBit;
#ifdef DEBUG_TOPP
      if (bid == BATCH_ID && tid == 0) {
        printf("targetIdx: %d, count_histogram[%d]: %d, current_sum: %f\n",
               targetIdx,
               targetIdx,
               count_histogram[targetIdx],
               current_sum);
      }
#endif
    }
    __syncthreads();
    constexpr int numPasses = calcNumPasses<T, BitsPerPass>();
    if constexpr (Pass != numPasses - 1) {
      for (int i = tid; i < NumBuckets; i += BlockSize) {
        histogram[i] = 0;
        count_histogram[i] = 0;
      }
    }
    if (tid == 0) {
      // recover
      counter->previousLen = current_len;
      counter->filterCnt = 0;
    }
    if constexpr (Pass == numPasses - 1) {
      const auto kthValueBits = counter->kthValueBits;
      const auto equal_value = twiddleOut<T>(kthValueBits, false);

      const T* last_data =
          out_buf ? out_buf : in_buf;  // 最后一次Pass的输入数据
      const int* last_idx_data = out_idx_buf ? out_idx_buf : in_idx_buf;
      const int last_len =
          out_buf ? current_len : counter->oriLen;  // 最后一次Pass的token长度
#ifdef DEBUG_TOPP
      if (bid == BATCH_ID && tid == 0) {
        printf("equal_value: %f, last_len: %d\n", equal_value, last_len);
      }
      __syncthreads();
#endif
      for (int i = tid; i < last_len; i += BlockSize) {
        if (last_data[i] == equal_value) {
          out[bid] = equal_value;
          ids[bid] = last_idx_data ? last_idx_data[i] : i;
        }
      }
    }
  }
}

template <typename T, int BitsPerPass>
__global__ void air_topp_init(Counter<T>* counters,
                              T* histograms,
                              int* countHistograms,
                              const T* in,
                              const T* ps,
                              hiprandState_t* curandstate,
                              const int bsz,
                              const int vocab_size,
                              const int buf_len,
                              const int num_buckets) {
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  Counter<T>* counter_now = counters + bid;
  T* histogram_now = histograms + bid * num_buckets;
  int* count_histogram_now = countHistograms + bid * num_buckets;
  const int offset = bid * vocab_size;
  if (tid == 0) {
    counter_now->in = in + offset;

    counter_now->len = vocab_size;
    counter_now->oriLen = vocab_size;
    counter_now->previousLen = vocab_size;

    const T p = ps[bid];
    const T rand_p = hiprand_uniform(curandstate + bid) * p;
    counter_now->p = rand_p;

    counter_now->sum = 0;

    counter_now->kthValueBits = 0;
    counter_now->filterCnt = 0;
    counter_now->finishedBlockCnt = 0;
  }
  for (int i = tid; i < num_buckets; i += blockDim.x) {
    histogram_now[i] = 0;
    count_histogram_now[i] = 0;
  }
}
#endif

template <typename T>
struct DataTypeTraits {
  using DataType = T;
};

template <>
struct DataTypeTraits<phi::dtype::float16> {
  using DataType = half;
};

#ifdef CUDA_BFLOAT16_AVAILABLE
template <>
struct DataTypeTraits<phi::dtype::bfloat16> {
  using DataType = __hip_bfloat16;
};
#endif

#define FINAL_MASK 0xFFFFFFFF

#define FIXED_BLOCK_DIM_BASE(dim, ...) \
  case (dim): {                        \
    constexpr auto kBlockDim = (dim);  \
    __VA_ARGS__;                       \
  } break

#ifdef PADDLE_WITH_HIP
#define WARP_SIZE 64
#define FIXED_BLOCK_DIM(...)                 \
  FIXED_BLOCK_DIM_BASE(1024, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_BASE(512, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(256, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(128, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(64, ##__VA_ARGS__);
#else
#define WARP_SIZE 32
#define FIXED_BLOCK_DIM(...)                 \
  FIXED_BLOCK_DIM_BASE(1024, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_BASE(512, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(256, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(128, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(64, ##__VA_ARGS__);   \
  FIXED_BLOCK_DIM_BASE(32, ##__VA_ARGS__)
#endif

struct SegmentOffsetIter {
  explicit SegmentOffsetIter(int num_cols) : num_cols_(num_cols) {}

  __host__ __device__ __forceinline__ int operator()(int idx) const {
    return idx * num_cols_;
  }

  int num_cols_;
};

template <typename T>
struct Pair {
  __device__ __forceinline__ Pair() {}
  __device__ __forceinline__ Pair(T value, int id) : v(value), id(id) {}

  __device__ __forceinline__ void set(T value, int id) {
    this->v = value;
    this->id = id;
  }

  __device__ __forceinline__ void operator=(const Pair<T>& in) {
    v = in.v;
    id = in.id;
  }

  __device__ __forceinline__ bool operator<(const T value) const {
    return (static_cast<float>(v) < static_cast<float>(value));
  }

  __device__ __forceinline__ bool operator>(const T value) const {
    return (static_cast<float>(v) > static_cast<float>(value));
  }
  __device__ __forceinline__ bool operator<(const Pair<T>& in) const {
    return (static_cast<float>(v) < static_cast<float>(in.v)) ||
           ((static_cast<float>(v) == static_cast<float>(in.v)) &&
            (id > in.id));
  }

  __device__ __forceinline__ bool operator>(const Pair<T>& in) const {
    return (static_cast<float>(v) > static_cast<float>(in.v)) ||
           ((static_cast<float>(v) == static_cast<float>(in.v)) &&
            (id < in.id));
  }

  T v;
  int id;
};

int GetBlockSize(int vocab_size) {
  if (vocab_size > 512) {
    return 1024;
  } else if (vocab_size > 256) {
    return 512;
  } else if (vocab_size > 128) {
    return 256;
  } else if (vocab_size > 64) {
    return 128;
  } else {
    return 64;
  }
}

inline int div_up(int a, int n) { return (a + n - 1) / n; }

template <typename T>
__device__ __forceinline__ void AddTo(Pair<T> topk[],
                                      const Pair<T>& p,
                                      int beam_size) {
  for (int k = beam_size - 2; k >= 0; k--) {
    if (topk[k] < p) {
      topk[k + 1] = topk[k];
    } else {
      topk[k + 1] = p;
      return;
    }
  }
  topk[0] = p;
}

template <typename T, int BlockSize>
__device__ __forceinline__ void GetTopK(
    Pair<T> topk[], const T* src, int idx, int dim, int beam_size) {
  while (idx < dim) {
    if (topk[beam_size - 1] < src[idx]) {
      Pair<T> tmp(src[idx], idx);
      AddTo<T>(topk, tmp, beam_size);
    }
    idx += BlockSize;
  }
}

template <typename T, int BlockSize>
__device__ __forceinline__ void GetTopK(Pair<T> topk[],
                                        const T* src,
                                        int idx,
                                        int dim,
                                        const Pair<T>& max,
                                        int beam_size) {
  while (idx < dim) {
    if (topk[beam_size - 1] < src[idx]) {
      Pair<T> tmp(src[idx], idx);
      if (tmp < max) {
        AddTo<T>(topk, tmp, beam_size);
      }
    }
    idx += BlockSize;
  }
}

template <typename T, int MaxLength, int BlockSize>
__device__ __forceinline__ void ThreadGetTopK(Pair<T> topk[],
                                              int* beam,
                                              int beam_size,
                                              const T* src,
                                              bool* firstStep,
                                              bool* is_empty,
                                              Pair<T>* max,
                                              int dim,
                                              const int tid) {
  if (*beam > 0) {
    int length = (*beam) < beam_size ? *beam : beam_size;
    if (*firstStep) {
      *firstStep = false;
      GetTopK<T, BlockSize>(topk, src, tid, dim, length);
    } else {
      for (int k = 0; k < MaxLength; k++) {
        if (k < MaxLength - (*beam)) {
          topk[k] = topk[k + *beam];
        } else {
          topk[k].set(std::numeric_limits<T>::min(), -1);
        }
      }
      if (!(*is_empty)) {
        GetTopK<T, BlockSize>(
            topk + MaxLength - *beam, src, tid, dim, *max, length);
      }
    }

    *max = topk[MaxLength - 1];
    if ((*max).id == -1) *is_empty = true;
    *beam = 0;
  }
}

template <typename T>
__forceinline__ __device__ Pair<T> WarpReduce(Pair<T> input) {
#pragma unroll
  for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1) {
    T tmp_val = phi::backends::gpu::CudaShuffleDownSync(
        FINAL_MASK, input.v, offset, WARP_SIZE);
    int tmp_id = phi::backends::gpu::CudaShuffleDownSync(
        FINAL_MASK, input.id, offset, WARP_SIZE);
    if (static_cast<float>(input.v) < static_cast<float>(tmp_val)) {
      input.v = tmp_val;
      input.id = tmp_id;
    }
  }
  return input;
}

template <typename T, int MaxLength, int BlockSize>
__device__ __forceinline__ void BlockReduce(Pair<T> shared_max[],
                                            Pair<T> topk[],
                                            Pair<T> beam_max[],
                                            int* beam,
                                            int* k,
                                            int* count,
                                            const int tid,
                                            const int wid,
                                            const int lane) {
  while (true) {
    __syncthreads();
    Pair<T> input_now = topk[0];
    input_now = WarpReduce(input_now);

    if (lane == 0) {
      shared_max[wid] = input_now;
    }
    __syncthreads();
    input_now = (tid < BlockSize / WARP_SIZE)
                    ? shared_max[lane]
                    : Pair<T>(std::numeric_limits<T>::min(), -1);
    if (wid == 0) {
      input_now = WarpReduce(input_now);
      if (lane == 0) shared_max[0] = input_now;
    }
    __syncthreads();
    if (tid == 0) {
      beam_max[*count] = shared_max[0];
      (*count)++;
    }
    int tid_max = shared_max[0].id % BlockSize;
    if (tid == tid_max) {
      (*beam)++;
    }
    if (--(*k) == 0) break;
    __syncthreads();

    if (tid == tid_max) {
      if (*beam < MaxLength) {
        topk[0] = topk[*beam];
      }
    }

    if (MaxLength < 5) {
      if (*beam >= MaxLength) break;
    } else {
#ifdef PADDLE_WITH_HIP
      uint64_t mask = 0u;
      mask = __ballot(true);
      if (tid_max / WARP_SIZE == wid) {
        if (__shfl_down(*beam, tid_max % WARP_SIZE, WARP_SIZE) == MaxLength)
          break;
      }
#else
      unsigned mask = 0u;
      mask = __ballot_sync(FINAL_MASK, true);
      if (tid_max / WARP_SIZE == wid) {
        if (__shfl_down_sync(
                FINAL_MASK, *beam, tid_max % WARP_SIZE, WARP_SIZE) == MaxLength)
          break;
      }
#endif
    }
  }
}

template <typename T>
__device__ inline T exponential_transform(T val, T lambda) {
#if defined(__NVCC__) || defined(__HIPCC__)
  T log = -std::numeric_limits<T>::epsilon() / 2;
  if (val < static_cast<T>(1.) - std::numeric_limits<T>::epsilon() / 2) {
    if (std::is_same<T, double>::value) {
      log = logf(val);
    } else {
      log = __logf(val);
    }
  }
  return static_cast<T>(-1.0) / lambda * log;
#else
  return static_cast<T>(-1.0) / lambda * std::log(static_cast<T>(1.0) - val);
#endif
}

template <typename T, int MaxLength, int TopPBeamTopK, int BlockSize>
__global__ void KeMatrixTopPBeamTopK(const T* src,
                                     const T* threshold,
                                     GPU(randState_t) * states,
                                     T* top_ps,
                                     int64_t* out_id,  // topk id
                                     T* out_val,       // topk val
                                     int64_t* topk_ids,
                                     T* topk_scores,
                                     int vocab_size,
                                     int* count_iter,
                                     int* count_iter_begin,
                                     const int k,
                                     const bool need_batch_random) {
  const int tid = threadIdx.x;
  const int wid = tid / WARP_SIZE;
  const int lane = tid % WARP_SIZE;
  const int bid = blockIdx.x;
  const float threshold_now =
      threshold ? static_cast<float>(threshold[bid]) : 0.f;

  int top_num = TopPBeamTopK;
  float top_p_num = static_cast<float>(top_ps[bid]);
  const int offset = bid * vocab_size;
  int64_t* topk_ids_now = nullptr;
  T* topk_scores_now = nullptr;
  if (k > 0) {
    topk_ids_now = topk_ids + bid * k;
    topk_scores_now = topk_scores + bid * k;
  }

  __shared__ Pair<T> shared_max[BlockSize / WARP_SIZE];
  __shared__ Pair<T> beam_max[TopPBeamTopK];

  Pair<T> topk[MaxLength];
  int beam = MaxLength;
  Pair<T> max;
  bool is_empty = false;
  bool firststep = true;
  __shared__ int count;

  if (tid == 0) {
    count = 0;
  }

  for (int j = 0; j < MaxLength; j++) {
    topk[j].set(std::numeric_limits<T>::min(), -1);
  }

  while (top_num) {
    ThreadGetTopK<T, MaxLength, BlockSize>(topk,
                                           &beam,
                                           TopPBeamTopK,
                                           src + offset,
                                           &firststep,
                                           &is_empty,
                                           &max,
                                           vocab_size,
                                           tid);
    BlockReduce<T, MaxLength, BlockSize>(
        shared_max, topk, beam_max, &beam, &top_num, &count, tid, wid, lane);
  }
  if (tid == 0) {
    count_iter_begin[bid] = count_iter[bid];
    float top_p = top_ps[bid];
    float sum_prob = 0.0f;
    bool flag = false;
    float max_val = 0.f;
    int max_id = -1;
    for (int i = 0; i < TopPBeamTopK; i++) {
      if (i < k) {
        topk_ids_now[i] = static_cast<int64_t>(beam_max[i].id);
        topk_scores_now[i] = beam_max[i].v;
      }
      if (!flag) {
        float val = static_cast<float>(beam_max[i].v);
        sum_prob += val;
        float random_ratio =
            exponential_transform(GPU(rand_uniform)(states + bid), 1.0f);
        float random_val = (val >= threshold_now ? val : 0.f) / random_ratio;
        if (max_val < random_val) {
          max_val = random_val;
          max_id = i;
        }
        if (sum_prob >= top_p) {
          flag = true;
          count_iter_begin[bid] += 1;
          if (max_id == -1) {
            // don't sample low score token
            out_id[bid] = static_cast<int64_t>(beam_max[0].id);
            out_val[bid] = beam_max[0].v;
          } else {
            out_id[bid] = static_cast<int64_t>(beam_max[max_id].id);
            out_val[bid] = beam_max[max_id].v;
          }
        }
      }
      if (flag && i >= k - 1) {
        break;
      }
    }
  }
}

template <typename T, int MaxLength, int TopPBeamTopK, int BlockSize>
__global__ void KeMatrixTopPBeamTopKFt(const T* src,
                                       const T* threshold,
                                       GPU(randState_t) * states,
                                       T* top_ps,
                                       int64_t* out_id,  // topk id
                                       T* out_val,       // topk val
                                       int64_t* topk_ids,
                                       T* topk_scores,
                                       int vocab_size,
                                       int* count_iter,
                                       int* count_iter_begin,
                                       const int k,
                                       const bool need_batch_random) {
  const int tid = threadIdx.x;
  const int wid = tid / WARP_SIZE;
  const int lane = tid % WARP_SIZE;
  const int bid = blockIdx.x;
  const float threshold_now =
      threshold ? static_cast<float>(threshold[bid]) : 0.f;

  int top_num = TopPBeamTopK;
  float top_p_num = static_cast<float>(top_ps[bid]);
  int64_t* topk_ids_now = nullptr;
  T* topk_scores_now = nullptr;
  if (k > 0) {
    topk_ids_now = topk_ids + bid * k;
    topk_scores_now = topk_scores + bid * k;
  }

  __shared__ Pair<T> shared_max[BlockSize / WARP_SIZE];
  __shared__ Pair<T> beam_max[TopPBeamTopK];

  Pair<T> topk[MaxLength];
  int beam = MaxLength;
  Pair<T> max;
  bool is_empty = false;
  bool firststep = true;
  __shared__ int count;

  if (tid == 0) {
    count = 0;
  }

  for (int j = 0; j < MaxLength; j++) {
    topk[j].set(std::numeric_limits<T>::min(), -1);
  }

  while (top_num) {
    ThreadGetTopK<T, MaxLength, BlockSize>(topk,
                                           &beam,
                                           TopPBeamTopK,
                                           src + bid * vocab_size,
                                           &firststep,
                                           &is_empty,
                                           &max,
                                           vocab_size,
                                           tid);
    BlockReduce<T, MaxLength, BlockSize>(
        shared_max, topk, beam_max, &beam, &top_num, &count, tid, wid, lane);
  }
  if (tid == 0) {
    count_iter_begin[bid] = count_iter[bid];
    float rand_top_p = GPU(rand_uniform)(states + bid) * top_p_num;
    top_ps[bid] = (T)rand_top_p;
    float sum_prob = 0.0f;
    bool flag = false;
    for (int i = 0; i < TopPBeamTopK; i++) {
      if (i < k) {
        topk_ids_now[i] = static_cast<int64_t>(beam_max[i].id);
        topk_scores_now[i] = beam_max[i].v;
      }
      if (!flag) {
        float val = static_cast<float>(beam_max[i].v);
        sum_prob += val;
#ifdef DEBUG_TOPP
        printf("bi: %d, top_p: %f, rand_top_p: %f, sum_prob: %f\n",
               bid,
               top_p_num rand_top_p,
               sum_prob);
#endif
        if (sum_prob >= rand_top_p) {
          flag = true;
          count_iter_begin[bid] += 1;
          if (val < threshold_now) {
            // don't sample low score token
            int start_id = i == 0 ? 0 : i - 1;
            for (int j = start_id; j >= 0; j--) {
              float val_now = static_cast<float>(beam_max[j].v);
              if (val_now >= threshold_now || j == 0) {
                out_id[bid] = static_cast<int64_t>(beam_max[j].id);
                out_val[bid] = beam_max[j].v;
                break;
              }
            }
          } else {
            out_id[bid] = static_cast<int64_t>(beam_max[i].id);
            out_val[bid] = beam_max[i].v;
          }
        }
      }
      if (flag && i >= k - 1) {
        break;
      }
    }
  }
}

__global__ void SetCountIter(int* count_iter, int num) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int idx = bid * blockDim.x + tid;
  for (int i = idx; i < num; i += gridDim.x * blockDim.x) {
    count_iter[i] = i;
  }
}

template <typename T>
__global__ void FillIndex(T* indices, T num_rows, T num_cols) {
  int col_id = threadIdx.x;
  int row_id = blockIdx.x;

  for (T j = row_id; j < num_rows; j += gridDim.x) {
    for (T i = col_id; i < num_cols; i += blockDim.x) {
      indices[j * num_cols + i] = i;
    }
  }
}

template <typename T, typename Context, int TopKMaxLength, int TopPBeamTopK>
void DispatchKeMatrixTopPBeamTopK(const Context& dev_ctx,
                                  const T* src,
                                  const T* threshold,
                                  GPU(randState_t) * states,
                                  T* top_ps,
                                  int64_t* out_id,  // topk id
                                  T* out_val,       // topk val
                                  int64_t* topk_ids,
                                  T* topk_scores,
                                  int vocab_size,
                                  int* count_iter,
                                  int* count_iter_begin,
                                  const int k,
                                  const int bs,
                                  const bool need_batch_random,
                                  const std::string& mode) {
  int BlockSize = GetBlockSize(vocab_size);
  if (mode == "truncated") {
    switch (BlockSize) {
      FIXED_BLOCK_DIM(
          KeMatrixTopPBeamTopKFt<T, TopKMaxLength, TopPBeamTopK, kBlockDim>
          <<<bs, kBlockDim, 0, dev_ctx.stream()>>>(src,
                                                   threshold,
                                                   states,
                                                   top_ps,
                                                   out_id,
                                                   out_val,
                                                   topk_ids,
                                                   topk_scores,
                                                   vocab_size,
                                                   count_iter,
                                                   count_iter_begin,
                                                   k,
                                                   need_batch_random));
      default:
        PD_THROW(
            "the input data shape has error in the topp_beam_topk kernel.");
    }
  } else {
    switch (BlockSize) {
      FIXED_BLOCK_DIM(
          KeMatrixTopPBeamTopK<T, TopKMaxLength, TopPBeamTopK, kBlockDim>
          <<<bs, kBlockDim, 0, dev_ctx.stream()>>>(src,
                                                   threshold,
                                                   states,
                                                   top_ps,
                                                   out_id,
                                                   out_val,
                                                   topk_ids,
                                                   topk_scores,
                                                   vocab_size,
                                                   count_iter,
                                                   count_iter_begin,
                                                   k,
                                                   need_batch_random));
      default:
        PD_THROW(
            "the input data shape has error in the topp_beam_topk kernel.");
    }
  }
}

struct BlockPrefixCallbackOp {
  // Running prefix
  float running_total;
  // Constructor
  __device__ BlockPrefixCallbackOp(float running_total)
      : running_total(running_total) {}
  // Callback operator to be entered by the first warp of threads in the block.
  // Thread-0 is responsible for returning a value for seeding the block-wide
  // scan.
  __device__ float operator()(float block_aggregate) {
    float old_prefix = running_total;
    running_total += block_aggregate;
    return old_prefix;
  }
};

template <typename T>
__device__ T max_func(const T a, const T b) {
  return a > b ? a : b;
}

template <typename T>
struct MaxOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const {
    return max_func(a, b);
  }
};

template <typename T, int BLOCK_SIZE>
__global__ void topp_sampling(T* sorted_probs,
                              int64_t* sorted_id,
                              T* out_val,
                              int64_t* out_id,
                              const T* top_ps,
                              const T* threshold,
                              GPU(randState_t) * states,
                              const int p_num,
                              const int vocab_size,
                              const bool need_batch_random,
                              int* count_iter,
                              int* count_iter_begin) {
  __shared__ int stop_shared;
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  constexpr int NUM_WARPS = BLOCK_SIZE / WARP_SIZE;
  const int lane_id = tid % WARP_SIZE;
  const int warp_id = tid / WARP_SIZE;
  const float p_t = static_cast<float>(top_ps[bid]);
  const float threshold_now =
      threshold ? static_cast<float>(threshold[bid]) : 0.f;
  if (tid == 0) {
    stop_shared = 0;
  }
  if (count_iter_begin[bid] == count_iter[bid + 1]) {
    // topk
    return;
  }

  typedef hipcub::BlockScan<float, BLOCK_SIZE> BlockScan;
  typedef hipcub::BlockReduce<Pair<T>, BLOCK_SIZE> BlockReduce;
  __shared__ typename BlockScan::TempStorage temp_storage;
  __shared__ typename BlockReduce::TempStorage temp_storage_reduce;

  // Initialize running total
  BlockPrefixCallbackOp prefix_op(0);

  int offset = bid * vocab_size;
#ifdef DEBUG_TOPP
  if (tid == 0) {
    printf(
        "first_elem1_1: %f, first_elem1_2: %f, first_id1_1: %d, first_id1_2: "
        "%d\n",
        static_cast<float>(sorted_probs[offset]),
        static_cast<float>(sorted_probs[offset + 1]),
        static_cast<int>(sorted_id[offset]),
        static_cast<int>(sorted_id[offset + 1]));
  }
#endif
  int end = ((vocab_size + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
  int i_activate = 0;
  float thread_offset = 0;
  Pair<T> max_thread_pair(static_cast<T>(0.), -1);
  for (int i = tid; i < end; i += BLOCK_SIZE) {
    float thread_count =
        (i < vocab_size) ? static_cast<float>(sorted_probs[offset + i]) : 0.f;
    BlockScan(temp_storage)
        .InclusiveSum(thread_count, thread_offset, prefix_op);

    if (thread_offset < p_t ||
        (thread_offset >= p_t && thread_offset - thread_count < p_t)) {
      float random_ratio =
          exponential_transform(GPU(rand_uniform)(states + bid), 1.0f);
      float tmp_val =
          (thread_count >= threshold_now ? thread_count : 0.f) / random_ratio;
      if (static_cast<float>(max_thread_pair.v) < tmp_val) {
        max_thread_pair.set(static_cast<T>(tmp_val), i);
      }
#ifdef DEBUG_TOPP
      if (i < 10) {
        printf(
            "tid: %d, i: %d, random_ratio: %f, thread_count: %f, tmp_val: %f, "
            "max_thread_pair.v: %f, max_thread_pair.id: %d\n",
            tid,
            i,
            random_ratio,
            thread_count,
            tmp_val,
            max_thread_pair.v,
            static_cast<int>(max_thread_pair.id));
      }
#endif
    }
#ifdef DEBUG_TOPP
    printf("tid: %d, thread_count: %f, thread_offset: %f\n",
           tid,
           thread_count,
           thread_offset);
#endif
#ifdef PADDLE_WITH_HIP
    uint64_t activate_mask = __ballot(p_t <= thread_offset);
#else
    uint32_t activate_mask = __ballot_sync(FINAL_MASK, p_t <= thread_offset);
#endif

    i_activate = i;
    if (activate_mask != 0) {
      if (lane_id == 0) {
        atomicAdd(&stop_shared, 1);
      }
    }
    __syncthreads();
    if (stop_shared > 0) {
      break;
    }
  }
  __syncthreads();
  if (stop_shared == 0) {
    if (tid == 0) {
      out_id[bid] = sorted_id[offset];
      out_val[bid] = sorted_probs[offset];
    }
    return;
  }

  Pair<T> max_pair = BlockReduce(temp_storage_reduce)
                         .Reduce(max_thread_pair, MaxOp<Pair<T>>());
  if (tid == 0) {
    if (max_pair.id == -1) {
      max_pair.id = 0;
    }
#ifdef DEBUG_TOPP
    printf("max_id: %d, max_val: %f\n",
           static_cast<int>(max_pair.id),
           static_cast<float>(max_pair.v));
#endif
    out_id[bid] = sorted_id[offset + max_pair.id];
    out_val[bid] = sorted_probs[offset + max_pair.id];
  }
}

template <typename T, int BLOCK_SIZE>
__global__ void topp_sampling_ft(T* sorted_probs,
                                 int64_t* sorted_id,
                                 T* out_val,
                                 int64_t* out_id,
                                 const T* top_ps,
                                 const T* threshold,
                                 GPU(randState_t) * states,
                                 const int p_num,
                                 const int vocab_size,
                                 const bool need_batch_random,
                                 int* count_iter,
                                 int* count_iter_begin) {
  __shared__ int stop_shared;
  __shared__ float rand_p;
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  constexpr int NUM_WARPS = BLOCK_SIZE / WARP_SIZE;
  const int lane_id = tid % WARP_SIZE;
  const int warp_id = tid / WARP_SIZE;
  const float p_t = static_cast<float>(top_ps[bid]);
  const float threshold_now =
      threshold ? static_cast<float>(threshold[bid]) : 0.f;
  if (tid == 0) {
    stop_shared = 0;
    rand_p = p_t;
#ifdef DEBUG_TOPP
    printf("bi: %d, p: %f\n", bid, rand_p);
#endif
  }
  if (count_iter_begin[bid] == count_iter[bid + 1]) {
    // topk
    return;
  }

  typedef hipcub::BlockScan<float, BLOCK_SIZE> BlockScan;
  typedef hipcub::BlockReduce<int, BLOCK_SIZE> BlockReduce;
  __shared__ typename BlockScan::TempStorage temp_storage;
  __shared__ typename BlockReduce::TempStorage temp_storage_reduce;
#ifdef PADDLE_WITH_HIP
  __shared__ uint64_t selected_shared[NUM_WARPS];
#else
  __shared__ uint32_t selected_shared[NUM_WARPS];
#endif
  int threshold_id = 0;

  // Initialize running total
  BlockPrefixCallbackOp prefix_op(0);

  if (lane_id == 0) {
    selected_shared[warp_id] = 0;
  }
  __syncthreads();

  int offset = bid * vocab_size;
#ifdef DEBUG_TOPP
  if (tid == 0) {
    printf(
        "first_elem1_1: %f, first_elem1_2: %f, first_id1_1: %d, first_id1_2: "
        "%d\n",
        static_cast<float>(sorted_probs[offset]),
        static_cast<float>(sorted_probs[offset + 1]),
        static_cast<int>(sorted_id[offset]),
        static_cast<int>(sorted_id[offset + 1]));
  }
#endif
  int end = ((vocab_size + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
  int i_activate = 0;
  float thread_offset = 0;
  for (int i = tid; i < end; i += BLOCK_SIZE) {
    float thread_count =
        (i < vocab_size) ? static_cast<float>(sorted_probs[offset + i]) : 0.f;
    if (i < vocab_size && thread_count >= threshold_now) {
      threshold_id = i;
    }
    BlockScan(temp_storage)
        .InclusiveSum(thread_count, thread_offset, prefix_op);

#ifdef PADDLE_WITH_HIP
    uint64_t activate_mask = __ballot(rand_p <= thread_offset);
#else
    uint32_t activate_mask = __ballot_sync(FINAL_MASK, rand_p <= thread_offset);
#endif

    i_activate = i;
    if (activate_mask != 0) {
      if (lane_id == 0) {
        atomicAdd(&stop_shared, 1);
        selected_shared[warp_id] = activate_mask;
      }
    }
    __syncthreads();
    if (stop_shared > 0) {
      break;
    }
  }
  __syncthreads();
  if (stop_shared == 0) {
    if (tid == 0) {
      out_id[bid] = sorted_id[offset];
      out_val[bid] = sorted_probs[offset];
#ifdef DEBUG_TOPP
      printf("stop_shared: %d, out_id: %d, out_val: %f\n",
             static_cast<int>(stop_shared),
             static_cast<int>(out_id[bid]),
             static_cast<float>(out_val[bid]));
#endif
    }
    return;
  }
#ifdef DEBUG_TOPP
  if (tid == 0) {
    printf(
        "first_elem2_1: %f, first_elem2_2: %f, first_id2_1: %d, first_id2_2: "
        "%d\n",
        static_cast<float>(sorted_probs[offset]),
        static_cast<float>(sorted_probs[offset + 1]),
        static_cast<int>(sorted_id[offset]),
        static_cast<int>(sorted_id[offset + 1]));
  }
#endif
  bool skip = (selected_shared[warp_id] > 0) ? false : true;
  for (int i = 0; i < warp_id; i++) {
    if (selected_shared[i] != 0) {
      // If the previous has stopped, skip the current warp
      skip = true;
    }
  }
  if (!skip) {
#ifdef PADDLE_WITH_HIP
    int active_lane_id =
        WARP_SIZE - __popcll(selected_shared[warp_id]);  // first not 0
#else
    int active_lane_id =
        WARP_SIZE - __popc(selected_shared[warp_id]);  // first not 0
#endif
    if (lane_id == active_lane_id) {
      float val = static_cast<float>(sorted_probs[offset + i_activate]);
#ifdef DEBUG_TOPP
      printf(
          "active_lane_id: %d, i_activate: %d.\n", active_lane_id, i_activate);
      for (int i = 0; i < active_lane_id; i++) {
        printf("p %d, value: %f\n",
               i,
               static_cast<float>(sorted_probs[offset + i]));
      }
#endif
      if (val < threshold_now) {
        // don't sample low score token
        int max_id =
            BlockReduce(temp_storage_reduce).Reduce(threshold_id, MaxOp<int>());
#ifdef PADDLE_WITH_HIP
        hiprandStatePhilox4_32_10_t rng;
        hiprand_init(bid * blockDim.x + tid, tid, 0, &rng);
        int random_id = hiprand(&rng) % (max_id + 1);
#else
        hiprandStatePhilox4_32_10_t rng;
        hiprand_init(bid * blockDim.x + tid, tid, 0, &rng);
        int random_id = hiprand(&rng) % (max_id + 1);
#endif
        out_id[bid] = sorted_id[offset + random_id];
        out_val[bid] = sorted_probs[offset + random_id];
      } else {
        out_id[bid] = sorted_id[offset + i_activate];
        out_val[bid] = sorted_probs[offset + i_activate];
      }
    }
  }
}

template <typename T, typename Context>
void DispatchTopPSampling(const Context& dev_ctx,
                          T* sorted_probs,
                          int64_t* sorted_id,
                          T* out_val,
                          int64_t* out_id,
                          const T* top_ps,
                          const T* threshold,
                          GPU(randState_t) * states,
                          const int p_num,
                          const int vocab_size,
                          const int bs,
                          const bool need_batch_random,
                          int* count_iter,
                          int* count_iter_begin,
                          const std::string& mode) {
  int BlockSize = GetBlockSize(vocab_size);
  if (mode == "truncated") {
    switch (BlockSize) {
      FIXED_BLOCK_DIM(
          topp_sampling_ft<T, kBlockDim>
          <<<bs, kBlockDim, 0, dev_ctx.stream()>>>(sorted_probs,
                                                   sorted_id,
                                                   out_val,
                                                   out_id,
                                                   top_ps,
                                                   threshold,
                                                   states,
                                                   p_num,
                                                   vocab_size,
                                                   need_batch_random,
                                                   count_iter,
                                                   count_iter_begin));
      default:
        PD_THROW("the input data shape has error in the topp_sampling kernel.");
    }
  } else {
    switch (BlockSize) {
      FIXED_BLOCK_DIM(
          topp_sampling<T, kBlockDim>
          <<<bs, kBlockDim, 0, dev_ctx.stream()>>>(sorted_probs,
                                                   sorted_id,
                                                   out_val,
                                                   out_id,
                                                   top_ps,
                                                   threshold,
                                                   states,
                                                   p_num,
                                                   vocab_size,
                                                   need_batch_random,
                                                   count_iter,
                                                   count_iter_begin));
      default:
        PD_THROW("the input data shape has error in the topp_sampling kernel.");
    }
  }
}

__global__ void setup_kernel(GPU(randState_t) * state,
                             int64_t* seed,
                             const int bs) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = idx; i < bs; i += gridDim.x * blockDim.x) {
    GPU(rand_init)(static_cast<uint64_t>(seed[i]), 0, 0, &state[i]);
  }
}

__global__ void setup_kernel(GPU(randState_t) * state,
                             const uint64_t seed,
                             const uint64_t offset,
                             const int bs,
                             const bool need_batch_random) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = idx; i < bs; i += gridDim.x * blockDim.x) {
    if (need_batch_random) {
      GPU(rand_init)(seed, i, offset, &state[i]);
    } else {
      GPU(rand_init)(seed, 0, offset, &state[i]);
    }
  }
}

template <typename T>
T* SafeGetTensorPtr(const DenseTensor& t) {
  return const_cast<T*>(t.data<T>());
}

template <typename T>
T* SafeGetTensorPtr(const DenseTensor* t) {
  return t ? SafeGetTensorPtr<T>(*t) : nullptr;
}

template <typename T>
T* SafeGetTensorPtr(const paddle::optional<DenseTensor>& t) {
  return t ? SafeGetTensorPtr<T>(t.get()) : nullptr;
}

template <typename T, typename Context>
void TopPSamplingKernel(const Context& dev_ctx,
                        const DenseTensor& x,
                        const DenseTensor& ps,
                        const paddle::optional<DenseTensor>& threshold,
                        const paddle::optional<DenseTensor>& topp_seed,
                        int seed,
                        int k,
                        const std::string& mode,
                        DenseTensor* out,
                        DenseTensor* ids,
                        DenseTensor* topk_scores,
                        DenseTensor* topk_ids) {
  typedef DataTypeTraits<T> traits_;
  typedef typename traits_::DataType DataType_;
  auto cu_stream = dev_ctx.stream();
  const auto* input = &x;
  // get the input dims
  const auto& in_dims = input->dims();
  int p_num = ps.numel();
  int bs = in_dims[0];
  int vocab_size = in_dims[1];
  T* out_ptr = dev_ctx.template Alloc<T>(out);
  int64_t* ids_ptr = dev_ctx.template Alloc<int64_t>(ids);
  T* topk_scores_data = nullptr;
  int64_t* topk_ids_data = nullptr;
  if (k > 0) {
    topk_scores_data = dev_ctx.template Alloc<T>(topk_scores);
    topk_ids_data = dev_ctx.template Alloc<int64_t>(topk_ids);
  }

  DenseTensor ps_now;
  ps_now.Resize(phi::make_ddim({bs, 1}));
  dev_ctx.template Alloc<T>(&ps_now);
  phi::Copy(dev_ctx, ps, dev_ctx.GetPlace(), false, &ps_now);

  DenseTensor inds_input;
  inds_input.Resize(phi::make_ddim({bs, vocab_size}));
  dev_ctx.template Alloc<int64_t>(&inds_input);

  DenseTensor sorted_out;
  sorted_out.Resize(phi::make_ddim({bs, vocab_size}));
  dev_ctx.template Alloc<T>(&sorted_out);

  DenseTensor sorted_id;
  sorted_id.Resize(phi::make_ddim({bs, vocab_size}));
  dev_ctx.template Alloc<int64_t>(&sorted_id);

  int BlockSize = GetBlockSize(vocab_size);

  switch (BlockSize) {
    FIXED_BLOCK_DIM(FillIndex<int64_t><<<bs, kBlockDim, 0, cu_stream>>>(
        inds_input.data<int64_t>(), bs, vocab_size));
    default:
      PD_THROW("the input data shape has error in the FillIndex kernel.");
  }
  int64_t* infer_seed = SafeGetTensorPtr<int64_t>(topp_seed);

  GPU(randState_t) * states{nullptr};
  phi::Allocator::AllocationPtr rand_states_buf{nullptr};
  rand_states_buf = phi::memory_utils::Alloc(
      dev_ctx.GetPlace(),
      bs * sizeof(GPU(randState_t)),
      phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
  states = reinterpret_cast<GPU(randState_t)*>(rand_states_buf->ptr());

  uint64_t seed_now = seed;
  uint64_t offset = 0;
  bool need_batch_random = false;

  if (infer_seed) {
    setup_kernel<<<1, 256, 0, cu_stream>>>(states, infer_seed, bs);
  } else {
    if (seed_now == -1) {
      need_batch_random = true;
      auto gen_cuda = dev_ctx.GetGenerator();
      uint64_t increment = ps.numel() * 4;
      auto seed_offset = gen_cuda->IncrementOffset(increment);
      seed_now = seed_offset.first;
      offset = seed_offset.second;
      setup_kernel<<<1, 256, 0, cu_stream>>>(
          states, seed_now, offset, bs, need_batch_random);
    } else {
      setup_kernel<<<1, 256, 0, cu_stream>>>(
          states, seed_now, offset, bs, need_batch_random);
    }
  }

  DenseTensor count_iter;
  count_iter.Resize(phi::make_ddim({bs + 1}));
  dev_ctx.template Alloc<int>(&count_iter);
  DenseTensor count_iter_begin;
  count_iter_begin.Resize(phi::make_ddim({bs}));
  dev_ctx.template Alloc<int>(&count_iter_begin);
  SetCountIter<<<1, 256, 0, cu_stream>>>(count_iter.data<int>(), bs + 1);

  T* threshold_data = SafeGetTensorPtr<T>(threshold);

  constexpr int TopKMaxLength = 2;
  constexpr int TopPBeamTopK = 20;

  DispatchKeMatrixTopPBeamTopK<T, Context, TopKMaxLength, TopPBeamTopK>(
      dev_ctx,
      x.data<T>(),
      threshold_data,
      states,
      ps_now.data<T>(),
      ids_ptr,
      out_ptr,
      topk_ids_data,
      topk_scores_data,
      vocab_size,
      count_iter.data<int>(),
      count_iter_begin.data<int>(),
      k,
      bs,
      need_batch_random,
      mode);
#if defined(PADDLE_WITH_CUDA) && CUDA_VERSION >= 12000
  if (FLAGS_use_air_topp) {
    static_assert(std::is_same<T, float>::value,
                  "air_topp only supports float now!");
    constexpr int BitsPerPass = 11;
    constexpr int SAMPLING_BLOCK_SIZE = 512;
    constexpr int INIT_BLOCK_SIZE = 1024;
    phi::Allocator::AllocationPtr counter_ptr{nullptr};
    counter_ptr = phi::memory_utils::Alloc(
        dev_ctx.GetPlace(),
        bs * sizeof(Counter<T>),
        phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
    Counter<T>* counters = reinterpret_cast<Counter<T>*>(counter_ptr->ptr());
    constexpr int numBuckets = calcNumBuckets<BitsPerPass>();
    const int buf_len = calcBufLen<T>(vocab_size);
    DenseTensor histograms, countHistograms, buf1, id_buf1, buf2, id_buf2;
    histograms.Resize(phi::make_ddim({bs, numBuckets}));
    countHistograms.Resize(phi::make_ddim({bs, numBuckets}));
    buf1.Resize(phi::make_ddim({bs, buf_len}));
    id_buf1.Resize(phi::make_ddim({bs, buf_len}));
    buf2.Resize(phi::make_ddim({bs, buf_len}));
    id_buf2.Resize(phi::make_ddim({bs, buf_len}));
    dev_ctx.template Alloc<T>(&histograms);
    dev_ctx.template Alloc<int>(&countHistograms);
    dev_ctx.template Alloc<T>(&buf1);
    dev_ctx.template Alloc<int>(&id_buf1);
    dev_ctx.template Alloc<T>(&buf2);
    dev_ctx.template Alloc<int>(&id_buf2);

    air_topp_init<T, BitsPerPass><<<bs, INIT_BLOCK_SIZE, 0, dev_ctx.stream()>>>(
        counters,
        histograms.data<T>(),
        countHistograms.data<int>(),
        x.data<T>(),
        ps.data<T>(),
        states,
        bs,
        vocab_size,
        buf_len,
        numBuckets);

    constexpr int VecSize = 16 / sizeof(T);

    const int max_block_num_vocab =
        ceilDiv(vocab_size, SAMPLING_BLOCK_SIZE * VecSize);
    auto kernel =
        air_topp_sampling<T, BitsPerPass, SAMPLING_BLOCK_SIZE, numBuckets, 0>;
    const int dev_id = 0;
    int sm_count;
    int act_blocks_per_sm;
    hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev_id);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &act_blocks_per_sm, kernel, SAMPLING_BLOCK_SIZE, 0);
    assert(act_blocks_per_sm > 1);
    const int block_per_wave = sm_count * act_blocks_per_sm;
    const int block_num_vocab =
        std::min(max_block_num_vocab, block_per_wave * 4 / bs);  // !!!
    VLOG(1) << "block_per_wave: " << block_per_wave
            << ", block_num_vocab: " << block_num_vocab;
    dim3 grid(block_num_vocab, bs);
    constexpr int numPasses = calcNumPasses<T, BitsPerPass>();
    for (int pass = 0; pass < numPasses; ++pass) {
      VLOG(1) << "pass: " << pass;
      if (pass == 0) {
        air_topp_sampling<T, BitsPerPass, SAMPLING_BLOCK_SIZE, numBuckets, 0>
            <<<grid, SAMPLING_BLOCK_SIZE, 0, dev_ctx.stream()>>>(
                counters,
                histograms.data<T>(),
                countHistograms.data<int>(),
                out_ptr,
                ids_ptr,
                buf1.data<T>(),
                id_buf1.data<int>(),
                buf2.data<T>(),
                id_buf2.data<int>(),
                count_iter.data<int>(),
                count_iter_begin.data<int>(),
                buf_len);
      } else if (pass == 1) {
        air_topp_sampling<T, BitsPerPass, SAMPLING_BLOCK_SIZE, numBuckets, 1>
            <<<grid, SAMPLING_BLOCK_SIZE, 0, dev_ctx.stream()>>>(
                counters,
                histograms.data<T>(),
                countHistograms.data<int>(),
                out_ptr,
                ids_ptr,
                buf1.data<T>(),
                id_buf1.data<int>(),
                buf2.data<T>(),
                id_buf2.data<int>(),
                count_iter.data<int>(),
                count_iter_begin.data<int>(),
                buf_len);
      } else if (pass == 2) {
        air_topp_sampling<T, BitsPerPass, SAMPLING_BLOCK_SIZE, numBuckets, 2>
            <<<grid, SAMPLING_BLOCK_SIZE, 0, dev_ctx.stream()>>>(
                counters,
                histograms.data<T>(),
                countHistograms.data<int>(),
                out_ptr,
                ids_ptr,
                buf1.data<T>(),
                id_buf1.data<int>(),
                buf2.data<T>(),
                id_buf2.data<int>(),
                count_iter.data<int>(),
                count_iter_begin.data<int>(),
                buf_len);
      } else {
        PD_THROW("pass must be 0,1 or 2!");
      }
    }
  } else {
#endif
    size_t temp_storage_bytes = 0;

    hipcub::TransformInputIterator<int, SegmentOffsetIter, int*>
        segment_offsets_t_begin(count_iter_begin.data<int>(),
                                SegmentOffsetIter(vocab_size));

    hipcub::TransformInputIterator<int, SegmentOffsetIter, int*>
        segment_offsets_t_end(count_iter.data<int>(),
                              SegmentOffsetIter(vocab_size));

    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        nullptr,
        temp_storage_bytes,
        reinterpret_cast<DataType_*>(const_cast<T*>(x.data<T>())),
        reinterpret_cast<DataType_*>(const_cast<T*>(sorted_out.data<T>())),
        inds_input.data<int64_t>(),
        sorted_id.data<int64_t>(),
        vocab_size * bs,
        bs,
        segment_offsets_t_begin,
        segment_offsets_t_end + 1,
        0,
        sizeof(T) * 8,
        cu_stream);

    temp_storage_bytes = div_up(temp_storage_bytes, 256) * 256;
    int64_t temp_size = temp_storage_bytes;
    DenseTensor temp_storage;
    temp_storage.Resize(phi::make_ddim({temp_size}));
    dev_ctx.template Alloc<uint8_t>(&temp_storage);

    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        temp_storage.data<uint8_t>(),
        temp_storage_bytes,
        reinterpret_cast<DataType_*>(const_cast<T*>(x.data<T>())),
        reinterpret_cast<DataType_*>(const_cast<T*>(sorted_out.data<T>())),
        inds_input.data<int64_t>(),
        sorted_id.data<int64_t>(),
        vocab_size * bs,
        bs,
        segment_offsets_t_begin,
        segment_offsets_t_end + 1,
        0,
        sizeof(T) * 8,
        cu_stream);

    DispatchTopPSampling<T>(dev_ctx,
                            sorted_out.data<T>(),
                            sorted_id.data<int64_t>(),
                            out_ptr,
                            ids_ptr,
                            ps_now.data<T>(),
                            threshold_data,
                            states,
                            p_num,
                            vocab_size,
                            bs,
                            need_batch_random,
                            count_iter.data<int>(),
                            count_iter_begin.data<int>(),
                            mode);
#if defined(PADDLE_WITH_CUDA) && CUDA_VERSION >= 12000
  }
#endif
}

}  // namespace phi

PD_REGISTER_KERNEL(
    top_p_sampling, GPU, ALL_LAYOUT, phi::TopPSamplingKernel, float) {}
