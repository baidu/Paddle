#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/aligned_vector.h"
#include "paddle/phi/kernels/fusion/gpu/fused_rope_utils.h"

namespace phi {
namespace fusion {

template <typename T, typename Context>
void FusedRopeKernel(const Context& dev_ctx,
                     const DenseTensor& q,
                     const paddle::optional<DenseTensor>& k,
                     const paddle::optional<DenseTensor>& v,
                     const paddle::optional<DenseTensor>& sin,
                     const paddle::optional<DenseTensor>& cos,
                     const paddle::optional<DenseTensor>& position_ids,
                     bool use_neox_rotary_style,
                     DenseTensor* out_q,
                     DenseTensor* out_k,
                     DenseTensor* out_v) {
  int64_t numel = q.numel();
  if (numel <= 0) return;
  dev_ctx.template Alloc<T>(out_q);

  // q.shape: [batch_size, seq_len, num_heads, head_dim]
  auto batch_size = q.dims()[0];
  auto seq_len = q.dims()[1];
  auto num_heads = q.dims()[2];
  auto head_dim = q.dims()[3];
  PADDLE_ENFORCE_EQ(head_dim % 2,
                    0,
                    phi::errors::InvalidArgument(
                        "The head_dim of input must be a multiple of 2."));

  constexpr const int vec_size = 2;

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, numel, vec_size);

  int64_t grid = config.block_per_grid.x;
  int64_t block = config.thread_per_block.x;
  auto stream = dev_ctx.stream();

  phi::Array<T*, 3> outs_data;
  phi::Array<const T*, 3> ins_data;
  phi::Array<const T*, 2> sin_cos_data;
  const int64_t* position_ids_data = NULL;

  ins_data[0] = q.data<T>();
  outs_data[0] = out_q->data<T>();
  int num_inputs = 0;

  if (k.get_ptr()) {
    dev_ctx.template Alloc<T>(out_k);
    ins_data[1] = k->data<T>();
    outs_data[1] = out_k->data<T>();
    num_inputs++;
  }

  if (v.get_ptr()) {
    dev_ctx.template Alloc<T>(out_v);
    ins_data[2] = v->data<T>();
    outs_data[2] = out_v->data<T>();
    num_inputs++;
  }

  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  MPType div_c = static_cast<MPType>(1.0f / head_dim);

  bool flag_sin_cos = false;

  if (sin.get_ptr() && cos.get_ptr()) {
    PADDLE_ENFORCE_EQ(sin.get_ptr()->dims(),
                      cos.get_ptr()->dims(),
                      phi::errors::InvalidArgument(
                          "The dims of sin and cos must be the same. But "
                          "recieved sin's dims is {%s}, cos's dims is {%s}.",
                          sin.get_ptr()->dims(),
                          cos.get_ptr()->dims()));

    auto sin_dims = sin.get_ptr()->dims();
    int dims_size = sin_dims.size();
    PADDLE_ENFORCE_EQ(
        (dims_size == 2 || dims_size == 4),
        true,
        phi::errors::InvalidArgument("The dims of sin and cos is expected to "
                                     "be 2 or 4, but recieved %d.",
                                     dims_size));
    if (dims_size == 4) {
      // sin.shape: [1, seq_len, 1, head_dim]
      PADDLE_ENFORCE_EQ(
          (sin_dims[0] == 1 && sin_dims[2] == 1),
          true,
          phi::errors::InvalidArgument(
              "The batch_size and num_heads of sin and cos must be 1."));
    }
    int sin_seq_len_dim = (dims_size) == 4 ? 1 : 0;

    if (position_ids.get_ptr()) {
      PADDLE_ENFORCE_EQ(
          (sin_dims[dims_size - 1] == head_dim &&
           sin_dims[sin_seq_len_dim] >= seq_len),
          true,
          phi::errors::InvalidArgument(
              "The seq_len of sin and cos must be greater than or equal to "
              "this of q. The head_dim of sin and cos must be the same as this "
              "of q. But recieved sin's "
              "shape is {%s}, q's shape is {%s}.",
              sin_dims,
              q.dims()));

      auto position_ids_dims = position_ids.get_ptr()->dims();
      PADDLE_ENFORCE_EQ(position_ids_dims.size(),
                        2,
                        phi::errors::InvalidArgument(
                            "The dims of position_ids is expected to "
                            "be 2, but recieved %d.",
                            position_ids_dims.size()));

      PADDLE_ENFORCE_EQ(
          (position_ids_dims[0] == batch_size &&
           position_ids_dims[1] == seq_len),
          true,
          phi::errors::InvalidArgument(
              "The batch_size and seq_len of position_ids must be the same as "
              "those of q. But recieved position_ids's "
              "shape is {%s}, q's shape is {%s}.",
              position_ids_dims,
              q.dims()));

      position_ids_data = position_ids->data<int64_t>();
    } else {
      PADDLE_ENFORCE_EQ(
          (sin_dims[dims_size - 1] == head_dim &&
           sin_dims[sin_seq_len_dim] == seq_len),
          true,
          phi::errors::InvalidArgument(
              "The seq_len and head_dim of sin and cos "
              "must be the same as those of q. But recieved sin's "
              "shape is {%s}, q's shape is {%s}.",
              sin_dims,
              q.dims()));
    }

    sin_cos_data[0] = sin->data<T>();
    sin_cos_data[1] = cos->data<T>();

    flag_sin_cos = true;
  }

  int sign = 1;
  if (use_neox_rotary_style) {
    VectorizedFusedRopeWithRotateEveryTwoKernel<T, MPType, vec_size>
        <<<grid, block, 0, stream>>>(ins_data,
                                     sin_cos_data,
                                     position_ids_data,
                                     flag_sin_cos,
                                     sign,
                                     batch_size,
                                     seq_len,
                                     num_heads,
                                     head_dim,
                                     outs_data,
                                     num_inputs,
                                     div_c);
  } else {
#ifdef __MUSACC__
  int32_t musa_batch_size[3] = {0};
  int32_t musa_seq_len[3] = {0};
  int32_t musa_num_heads[3] = {0};
  int32_t musa_head_dim[3] = {0};
  musa_batch_size[0] = q.dims()[0];
  musa_seq_len[0] = q.dims()[1];
  musa_num_heads[0] = q.dims()[2];
  musa_head_dim[0] = q.dims()[3];
  if(k.get_ptr()) {
    musa_batch_size[1] = k->dims()[0];
    musa_num_heads[1] = k->dims()[2];
    musa_seq_len[1] = k->dims()[1];
    musa_head_dim[1] = k->dims()[3];
  }
  if(v.get_ptr()) {
    musa_batch_size[2] = v->dims()[0];
    musa_num_heads[2] = v->dims()[2];
    musa_seq_len[2] = v->dims()[1];
    musa_head_dim[2] = v->dims()[3];
  }

  if(flag_sin_cos && (std::is_same<T, float16>::value || std::is_same<T, half>::value) && musa_head_dim[0]%16 == 0 && musa_head_dim[1]%16 == 0 && musa_head_dim[2]%16 == 0) {
    const int32_t v_len = 8;
    for(int i = 0;i <= num_inputs; i++) {
      const int32_t block_dim_x = musa_head_dim[i] / (2*v_len);
      const int32_t tile_s = (512 + block_dim_x - 1) / block_dim_x;

      half *musa_input_data = (half *)ins_data[i];
      half *musa_output_data = (half *)outs_data[i];
      half *sin_data = (half *)sin_cos_data[0];
      half *cos_data =  (half *)sin_cos_data[1];
      const int64_t in_stride_b = musa_seq_len[i] * musa_num_heads[i] * musa_head_dim[i];
      const int64_t in_stride_s = musa_num_heads[i] * musa_head_dim[i];
      const int64_t in_stride_h = musa_head_dim[i];
      const int64_t pos_stride_b = musa_seq_len[i];

      dim3 musa_block(block_dim_x, tile_s, 1);
      dim3 musa_grid((musa_seq_len[i]+tile_s-1)/tile_s, musa_batch_size[i], 1);

      // vec_size = 8, bwd = false
      fusedRopeInterleaved<8, false>
            <<<musa_grid, musa_block, 0, stream>>>(musa_output_data,
                                                   musa_input_data, 
                                                   cos_data, 
                                                   sin_data, 
                                                   position_ids_data, 
                                                   musa_batch_size[i],
                                                   musa_seq_len[i],
                                                   musa_num_heads[i], 
                                                   musa_head_dim[i], 
                                                   in_stride_b, in_stride_s, in_stride_h, 
                                                   in_stride_b, in_stride_s, in_stride_h, 
                                                   pos_stride_b);
    }
  } else {
#endif
    VectorizedFusedRopeWithRotateHalfKernel<T, MPType, vec_size>
        <<<grid, block, 0, stream>>>(ins_data,
                                     sin_cos_data,
                                     position_ids_data,
                                     flag_sin_cos,
                                     sign,
                                     batch_size,
                                     seq_len,
                                     num_heads,
                                     head_dim,
                                     outs_data,
                                     num_inputs,
                                     div_c);
#ifdef __MUSACC__
    } // musa kernel else
#endif
  }
}
}  // namespace fusion
}  // namespace phi

PD_REGISTER_KERNEL(fused_rotary_position_embedding,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::FusedRopeKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16){};
